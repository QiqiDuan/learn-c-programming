#include "hip/hip_runtime.h"
#include <stdio.h>

#include "cuda_utils_opal.h"



__device__ int d_increase = 0;

__global__ void d_printf(void) {
    int print_id;
    print_id = atomicAdd(&d_increase, 1);
    printf("Block %d && Thread %d : %d\n",
        blockIdx.x, threadIdx.x, print_id);
}



int main( void ) {
    printf("* start test:\n");
    d_printf<<<3, 8>>>();
    HCE(hipDeviceSynchronize());
    HCE(hipGetLastError());
    printf("* end test!\n");

    return EXIT_SUCCESS;
}
