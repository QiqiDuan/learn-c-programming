#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <unistd.h>
#include <time.h>

#include "cuda_utils_opal.h"



typedef float FT;

__global__ void d_sum_vect(FT *vect_a, FT *vect_b, FT *vect_c, 
        const unsigned int vect_size) {
    unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    if(tidx < vect_size) {
        vect_c[tidx] = vect_a[tidx] + vect_b[tidx];
    }
}

void sum_vect(FT *vect_a, FT *vect_b, FT *vect_c, 
        const unsigned int vect_size) {
    for(unsigned int vect_ind = 0; vect_ind < vect_size; vect_ind++) {
        vect_c[vect_ind] = vect_a[vect_ind] + vect_b[vect_ind];
    }
}

int check_sum(FT *vect_a, FT *vect_b, const unsigned int vect_size) {
    for(unsigned int vect_ind = 0; vect_ind < vect_size; vect_ind++) {
        if(fabs(vect_a[vect_ind] - vect_b[vect_ind]) > DBL_EPSILON) {
            printf("\nERROR :: %lf vs. %lf.\n",
                    (double) vect_a[vect_ind], (double) vect_b[vect_ind]);
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}

void init_vect(FT *vect, const unsigned int vect_size) {
    srand((unsigned int) time(NULL));
    for(unsigned int vect_ind = 0; vect_ind < vect_size; vect_ind++) {
        vect[vect_ind] = (FT) rand() / RAND_MAX;
    }
} 

void print_vect_samples(FT *vect_a, FT *vect_b, const unsigned int vect_size) {
    for(unsigned int vect_ind = 0; vect_ind < vect_size; 
            vect_ind += (vect_size / 10)) {
        printf("vect_a[%d] = %lf && vect_b[%d] = %lf.\n",
            vect_ind, vect_a[vect_ind], vect_ind, vect_b[vect_ind]);
    }
}



int main( void ) {
    const int dev_ind = 0;
    HCE(hipSetDevice(dev_ind));

    const int trial_num = 2000;

    const unsigned int vect_size = 1 << 25;
    const unsigned int vect_size_bytes = sizeof(FT) * vect_size;
    printf("Vector Size = %d.\n", vect_size);
    FT *vect_a, *vect_b, *vect_c, *vect_c_cpy_gpu;
    vect_a = (FT *) malloc(vect_size_bytes);
    vect_b = (FT *) malloc(vect_size_bytes);
    vect_c = (FT *) malloc(vect_size_bytes);
    vect_c_cpy_gpu = (FT *) malloc(vect_size_bytes);

    init_vect(vect_a, vect_size);
    init_vect(vect_b, vect_size);

    time_t cpu_time_start, cpu_time_end;
    time(&cpu_time_start);
    for(unsigned int trial_ind = 0; trial_ind < trial_num; trial_ind++) {
        sum_vect(vect_a, vect_b, vect_c, vect_size);
    }
    time(&cpu_time_end);
    double cpu_time = difftime(cpu_time_end, cpu_time_start);
    printf("CPU time : %7.2lf.\n", cpu_time);

    FT *d_vect_a, *d_vect_b, *d_vect_c;
    HCE(hipMalloc((FT **) &d_vect_a, vect_size_bytes));
    HCE(hipMalloc((FT **) &d_vect_b, vect_size_bytes));
    HCE(hipMalloc((FT **) &d_vect_c, vect_size_bytes));

    time_t gpu_time_start, gpu_time_end;
    time(&gpu_time_start);
    HCE(hipMemcpy(d_vect_a, vect_a, vect_size_bytes, hipMemcpyHostToDevice));
    HCE(hipMemcpy(d_vect_b, vect_b, vect_size_bytes, hipMemcpyHostToDevice));
    dim3 block(1024);
    dim3 grid((vect_size + block.x - 1) / block.x);
    for(unsigned int trial_ind = 0; trial_ind < trial_num; trial_ind++) {
        d_sum_vect<<<grid, block>>>(d_vect_a, d_vect_b, d_vect_c, 
            vect_size_bytes);
        HCE(hipDeviceSynchronize());
        HCE(hipGetLastError());
    }
    time(&gpu_time_end);
    double gpu_time = difftime(gpu_time_end, gpu_time_start);
    printf("GPU time : %7.2lf.\n", gpu_time);

    HCE(hipMemcpy(vect_c_cpy_gpu, d_vect_c, vect_size_bytes, 
                hipMemcpyDeviceToHost));

    check_sum(vect_c_cpy_gpu, vect_c, vect_size);
    print_vect_samples(vect_c_cpy_gpu, vect_c, vect_size);
    
    HCE(hipFree(d_vect_a));
    HCE(hipFree(d_vect_b));
    HCE(hipFree(d_vect_c));

    free(vect_a);
    free(vect_b);
    free(vect_c);
    free(vect_c_cpy_gpu);

    HCE(hipDeviceReset());
    return EXIT_SUCCESS;
}
