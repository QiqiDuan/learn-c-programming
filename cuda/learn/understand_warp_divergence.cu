#include "hip/hip_runtime.h"
/* ***************************************************************************** *
 * Understand Warp Divergence.
 *
 * Version: v00.00.000 [2017-08-01 19:30:00]
 * Author : DQQ-077    [Qiqi Duan]
 *          SC         [Chang Shao]
 * Email  : duanqq@mail.sustc.edu.cn
 *          shaoc@mail.sustc.edu.cn
 * OUTPUT:
 * -------
 *  # nvcc understand_warp_divergence.cu
 *      Run time :: diverge_warp() vs. avoid_diverge_warp()
 *                | 0.00157   vs. 0.00090
 *  # nvcc -O3 understand_warp_divergence.cu
 *      Run time :: diverge_warp() vs. avoid_diverge_warp()
 *                | 0.00157   vs. 0.00089
 *  # nvcc -g -G understand_warp_divergence.cu
 *      Run time :: diverge_warp() vs. avoid_diverge_warp()
 *                | 0.01510   vs. 0.02559
 * ***************************************************************************** */
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include "cuda_utils_opal.h"

#define THREAD_NUM_IN_WARP 32

__global__ void diverge_warp(double *array) {
    double a = 2.2, b = 5.5;
    size_t tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if(tidx % 3 == 0) {
        array[tidx] = a + b;
    } else if(tidx % 3 == 1) {
        array[tidx] = a * b;
    } else {
        array[tidx] = a / b;
    }
}

__global__ void avoid_diverge_warp(double *array) {
    double a = 2.2, b = 5.5;
    size_t tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if(tidx / THREAD_NUM_IN_WARP % 3 == 0) {
        array[tidx] = a + b;
    } else if(tidx / THREAD_NUM_IN_WARP % 3 == 1) {
        array[tidx] = a * b;
    } else {
        array[tidx] = a / b;
    }
}

int main(void) {
    printf("Understand Warp Divergence ::\n");
    HCE(hipSetDevice(0));
    double run_time_diverge_warp = 0.0,
           run_time_avoid_diverge_warp = 0.0;

    struct timeval run_time_start, run_time_end;
    double run_time_start_double, run_time_end_double;

    size_t array_length = 1 << 24;
    size_t array_length_bytes = array_length * sizeof(double);
    double *array1, *array2;
    array1 = (double *)malloc(array_length_bytes);
    array2 = (double *)malloc(array_length_bytes);
    double *d_array1, *d_array2;
    HCE(hipMalloc((double **)&d_array1, array_length_bytes));
    HCE(hipMalloc((double **)&d_array2, array_length_bytes));
    HCE(hipDeviceSynchronize());
    HCE(hipGetLastError());

    dim3 block_size(1024);
    dim3 grid_size((array_length + block_size.x - 1) / block_size.x);

    gettimeofday(&run_time_start, NULL);
    avoid_diverge_warp<<<grid_size, block_size>>>(d_array2);
    HCE(hipDeviceSynchronize());
    HCE(hipGetLastError());
    gettimeofday(&run_time_end, NULL);
    HCE(hipMemcpy(array2, d_array2, array_length_bytes, hipMemcpyDeviceToHost));
    run_time_start_double = (double)run_time_start.tv_sec
        + (double)run_time_start.tv_usec / 1e6;
    run_time_end_double = (double)run_time_end.tv_sec
        + (double)run_time_end.tv_usec / 1e6;
    run_time_avoid_diverge_warp = run_time_end_double - run_time_start_double;

    gettimeofday(&run_time_start, NULL);
    diverge_warp<<<grid_size, block_size>>>(d_array1);
    HCE(hipDeviceSynchronize());
    HCE(hipGetLastError());
    gettimeofday(&run_time_end, NULL);
    HCE(hipMemcpy(array1, d_array1, array_length_bytes, hipMemcpyDeviceToHost));
    run_time_start_double = (double)run_time_start.tv_sec
        + (double)run_time_start.tv_usec / 1e6;
    run_time_end_double = (double)run_time_end.tv_sec
        + (double)run_time_end.tv_usec / 1e6;
    run_time_diverge_warp = run_time_end_double - run_time_start_double;

    printf("Run time :: diverge_warp() vs. avoid_diverge_warp()\n");
    printf("          | %-9.5lf vs. %-9.5lf\n",
            run_time_diverge_warp, run_time_avoid_diverge_warp);

    printf("\n\n\nSampling:\n");
    for(size_t i = 0; i < 4; i++) {
        printf("%zu :: %5.4lf vs. %5.4lf\n", i, array1[i], array2[i]);
    }
    printf("...\n");
    for(size_t i = array_length - 4; i < array_length; i++) {
        printf("%zu :: %5.4lf vs. %5.4lf\n", i, array1[i], array2[i]);
    }

    hipFree(d_array1);
    hipFree(d_array2);
    free(array1);
    free(array2);
}
