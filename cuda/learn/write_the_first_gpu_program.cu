#include "hip/hip_runtime.h"
/* ***************************************************************************** *
 * Print Message From GPU on CPU.
 *
 * Version: v00.00.000 [2017-07-14 11:30:00]
 * Author : DQQ-077    [Qiqi Duan]
 *          SC         [Chang Shao]
 * Email  : duanqq@mail.sustc.edu.cn
 *          shaoc@mail.sustc.edu.cn
 *
 * OUTPUT:
 * -------

 Write the first GPU program!


 Write the second GPU program!
 Hello World!
 Hello World!
 Hello World!
 00 :: Hello World!
 01 :: Hello World!
 02 :: Hello World!
 03 :: Hello World!
 04 :: Hello World!
 05 :: Hello World!
 06 :: Hello World!
 07 :: Hello World!
 08 :: Hello World!
 09 :: Hello World!
 10 :: Hello World!
 11 :: Hello World!
 12 :: Hello World!
 13 :: Hello World!
 14 :: Hello World!
 15 :: Hello World!
 16 :: Hello World!
 17 :: Hello World!
 18 :: Hello World!
 19 :: Hello World!
 20 :: Hello World!
 21 :: Hello World!
 22 :: Hello World!
 23 :: Hello World!
 24 :: Hello World!
 25 :: Hello World!
 26 :: Hello World!
 27 :: Hello World!
 28 :: Hello World!
 29 :: Hello World!
 30 :: Hello World!
 31 :: Hello World!


 Write the third GPU program!
 BlockIdx 0 && threadIdx 0 :: 0
 BlockIdx 0 && threadIdx 1 :: 2
 BlockIdx 0 && threadIdx 2 :: 4
 BlockIdx 1 && threadIdx 0 :: 6
 BlockIdx 1 && threadIdx 1 :: 8
 BlockIdx 1 && threadIdx 2 :: 10


 Write the fourth GPU program!
 * ***************************************************************************** */
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils_opal.h"



__global__ void print_hello_world(void) {
    printf("Hello World!\n");
}

__global__ void print_hello_world2(void) {
    printf("%02d :: Hello World!\n", threadIdx.x);
}

__device__ int show_base = 0;
__global__ void print_atomicAdd(void) {
    int show_id;
    show_id = atomicAdd(&show_base, 2);
    printf("BlockIdx %d && threadIdx %d :: %d\n",
            blockIdx.x, threadIdx.x, show_id);
}

int main(void) {
    printf("Write the first GPU program!\n");
    print_hello_world<<<1, 3>>>();
    printf("\n\n");

    printf("Write the second GPU program!\n");
    print_hello_world2<<<1, 32>>>();
    HCE(hipDeviceSynchronize());
    HCE(hipGetLastError());
    printf("\n\n");

    printf("Write the third GPU program!\n");
    print_atomicAdd<<<2, 3>>>();
    HCE(hipDeviceSynchronize());
    HCE(hipGetLastError());
    printf("\n\n");

    printf("Write the fourth GPU program!\n");
    print_hello_world2<<<1, 32>>>(); // cannot work! WHY?

    return EXIT_SUCCESS;
}
